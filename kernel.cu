#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void norm (float *d_Input, float *d_Output, int n) {

    // the column to be computed
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int row, mu, sigma;

    // the same alogrithm as sequential since the computation does not depend on rows
    if (col < n){
        mu = (float)0.0;
        for (row=0; row < n; row++){
            mu += d_Input[col*n+row];
        }
        mu /= (float) n;

        __syncthreads();

        sigma = (float)0.0;
        for (row=0; row < n; row++){

            sigma += powf(d_Input[col*n+row] - mu, (float)2.0);
        }
        sigma /= (float) n;

        __syncthreads();

        for (row=0; row < n; row++) {
            if (sigma == (float)0.0){
                d_Output[row*n+col] = (float)0.0;
            }
            else{
                d_Output[row*n+col] = (d_Input[col*n+row] - mu) / sigma;
            }
        }
    }
}