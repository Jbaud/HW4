#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void norm (float *d_Input, float *d_Output, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int row, mu, sigma;
    if (col < n){
        mu = (float)0.0;
        for (row=0; row < n; row++)
            mu += d_Input[col*n+row];
        mu /= (float) n;

        __syncthreads();

        sigma = (float)0.0;
        for (row=0; row < n; row++)
            sigma += powf(d_Input[col*n+row] - mu, (float)2.0);
        sigma /= (float) n;

        __syncthreads();

        for (row=0; row < n; row++) {
            if (sigma == (float)0.0)
                d_Output[row*n+col] = (float)0.0;
            else
                d_Output[row*n+col] = (d_Input[col*n+row] - mu) / sigma;
        }
    }
}